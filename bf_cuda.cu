#include "hip/hip_runtime.h"
#ifndef __HIPCC__  
    #define __HIPCC__
#endif
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <iostream>
#include <fstream>
#include <sstream>
#include <time.h>
#include "kernel.h"

#define Nsample 1000
#define Nchan  128
#define Frame 100
#define NUM 128000
//#define NUM 89600
using namespace std;
__constant__ const float startDepth = 0.0;
__constant__ const float fs = 20.0;
__constant__ const float fc = 5.0;
__constant__ const float soundv = 1.54;
__constant__ const float lambda = 0.3;
__constant__ const float pitch =  0.3;
__constant__ const float dz =  0.077; //mm
__constant__ const float f_num = 3.0;

__global__ void delayAndSum(float *input, float *out, float* x_axis, float* z_axis, float* toff , float *apod){		
	float idx;
	int idx_floor;
	float delay;
	int index = blockIdx.y * NUM + blockIdx.x * blockDim.x + threadIdx.x; // 2d 
	float s = 0.0f;
	float point = 0.0f;
	float point_next = 0.0f;
	for(int i = 0; i < Nchan; i++){
		delay = (((sqrtf( z_axis[blockIdx.x * blockDim.x + threadIdx.x] * z_axis[blockIdx.x * blockDim.x + threadIdx.x] + (x_axis[i*blockDim.x+threadIdx.x] - x_axis[blockIdx.x * blockDim.x + threadIdx.x]) * (x_axis[i*blockDim.x+threadIdx.x] - x_axis[blockIdx.x * blockDim.x + threadIdx.x]))))+z_axis[blockIdx.x * blockDim.x + threadIdx.x]) / soundv;
		idx = ( (delay - (*toff)*1000000)*fs );
		if( (idx >= 0) && (idx < Nsample - 1) ){
			idx_floor = floorf((idx));  
			point = input[blockIdx.y * NUM + i * blockDim.x + idx_floor];
			point_next = input[blockIdx.y * NUM + i * blockDim.x + idx_floor +1];

			if ( fabs(x_axis[i*blockDim.x + threadIdx.x] - x_axis[blockIdx.x * blockDim.x + threadIdx.x]) <= z_axis[blockIdx.x * blockDim.x + threadIdx.x] / f_num ){		
				s += (  (float)(point_next - point) * (idx - (float)idx_floor) + point ) ;

			}
		}   
		out[index] = s;
	}

}

extern "C" int bf2(float* input, float* out, float *z_axis, float *x_axis, float *toff, float *apod){
	hipError_t cudaStatus;
	float *input_d, *out_d, *z_axis_d, *x_axis_d, *toff_d, *apod_d;
	cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
		hipDeviceReset();
		return 1;      
    }
    cudaStatus = hipMalloc((void**)&input_d, NUM*Frame * sizeof(float));
    if (cudaStatus != hipSuccess) {
		hipDeviceReset();
        return 2;
    }
    cudaStatus = hipMalloc((void**)&out_d, NUM*Frame * sizeof(float));
    if (cudaStatus != hipSuccess) {
		hipDeviceReset();
        return 3;
    }
	cudaStatus = hipMalloc((void**)&z_axis_d, NUM * sizeof(float));
    if (cudaStatus != hipSuccess) {
		hipDeviceReset();
         return 4;
    }
	cudaStatus = hipMalloc((void**)&x_axis_d, NUM* sizeof(float));
    if (cudaStatus != hipSuccess) {
		hipDeviceReset();
         return 5;
    }

	cudaStatus = hipMalloc((void**)&toff_d, 1*sizeof(float));
    if (cudaStatus != hipSuccess) {
		hipDeviceReset();
         return 6;
    }
	cudaStatus = hipMalloc((void**)&apod_d, Nchan*sizeof(float));
    if (cudaStatus != hipSuccess) {
		hipDeviceReset();
         return 7;
    }
	hipMemset(out_d,0,NUM*Frame*sizeof(float));
	hipMemset(z_axis_d,0,NUM*sizeof(float));
	hipMemset(x_axis_d,0,NUM*sizeof(float));
	
	cudaStatus = hipMemcpy(out_d, out, NUM * Frame * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
		hipDeviceReset();
         return 11;
    }
	cudaStatus = hipMemcpy(z_axis_d, z_axis, NUM  * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
		hipDeviceReset();
         return 12;
    }
	cudaStatus = hipMemcpy(x_axis_d, x_axis, NUM  * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
         return 13;
    }

	cudaStatus = hipMemcpy(input_d, input, NUM * Frame * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
		hipDeviceReset();
        return 14; 
    }
	cudaStatus = hipMemcpy(toff_d, toff, sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
		hipDeviceReset();
        return 15; 
    }
	cudaStatus = hipMemcpy(apod_d, apod, Nchan*sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
		hipDeviceReset();
        return 16; 
    }
	dim3 bb(Nchan,Frame);
	delayAndSum<<< bb, Nsample>>>(input_d,out_d,x_axis_d,z_axis_d,toff_d,apod_d);

	cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
		hipDeviceReset();
        return 10; 
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
		hipDeviceReset();
       return 11;  
    }

	cudaStatus = hipMemcpy(out, out_d, NUM * Frame * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
		hipDeviceReset();
        return 12; 
    }

	hipFree(input_d);
	hipFree(out_d);
	hipFree(x_axis_d);
	hipFree(z_axis_d);
	hipFree(toff_d);
	hipFree(apod_d);
	hipDeviceReset();
	return 0;
}





